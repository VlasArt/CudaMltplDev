﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <time.h>
#include <hiprand/hiprand.h>

bool stream_init(hipStream_t* stream)
{
    int* num = new int;
    hipGetDeviceCount(num);

    try {
        for (int i = 0;i < *num;i++) {
            hipStreamCreate(&stream[i]);
        }
    }
    catch (...) {
        return false;
    }

    delete num;
    return true;
}

bool stream_dispose(hipStream_t* stream)
{
    int* num = new int;
    hipGetDeviceCount(num);

    try {
        for (int i = 0;i < *num;i++) {
            hipStreamDestroy(stream[i]);
        }
    }
    catch (...) {
        return false;
    }

    delete num;
    return true;
}

bool generateRandArray(float* numArray, int arraySize)
{
    try {
        float* dev_a;
        srand(time(NULL));

        hipSetDevice(0);
        hipMalloc((void**)&dev_a, arraySize * sizeof(int));

        hiprandGenerator_t gen;
        hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_DEFAULT);
        hiprandSetPseudoRandomGeneratorSeed(gen, 1234ULL);
        hiprandGenerateUniform(gen, dev_a, arraySize);
        hipMemcpy(numArray, dev_a, arraySize * sizeof(int), hipMemcpyDeviceToHost);

        hipFree(dev_a);
        hiprandDestroyGenerator(gen);
    }
    catch (...) {
        return false;
    }
    return true;
}

bool askAboutMemory(int arraySize)
{
    char ch;
    
    std::cout << "\nThis programm need more then " << arraySize * sizeof(float) / 1024. / 1024 / 1024 * 2 << " GB RAM. Continue?\n(Y/N)_";
    std::cin >> ch;
    std::cout << std::endl;

    if (ch != 'y' && ch != 'Y')
        return false;

    return true;
}

void lazzzyArrayPrint(float* arr, int arrSize)
{
    std::cout << '\n' << "Array len: " << arrSize << '\n' << "Array items:" << '\n' << std::endl;
    std::cout << arr[0]<< " " << arr[1] << " " << arr[2] << std::endl; 
    std::cout <<  " ... " << std::endl;
    std::cout << arr[arrSize / 2 - 1] << " " << arr[arrSize/2] << " " << arr[arrSize / 2 + 1] << std::endl;
    std::cout << " ... " << std::endl;
    std::cout << arr[arrSize - 3] << " " << arr[arrSize - 2] << " " << arr[arrSize - 1] << '\n' << std::endl;
}

__global__ void _kernel(float* a)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    int d;
    
      
    hipGetDevice(&d);
    printf("Kernel %d working\n", d);
    hipDeviceGetAttribute(&d, hipDeviceAttributePciBusId , d);
    a[i] = a[i] * a[i] + d;
}

int main()
{
    // Разминочный блок. Определяет количество гпу и выводит их характеристики    
    int num;
    hipGetDeviceCount(&num);
    std::cout << "Detcted device count: " << num << '\n' << std::endl;

    for (int i = 0;i < num;i++) {
        // Query the device properties.
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        std::cout << "Device id: " << i << std::endl;
        std::cout << "Device name: " << prop.name << std::endl;
    }

    // Массив потоков. Каждый поток - это отдельная видеокарта.
    // Порядковый номер в массиве - id карты
    hipStream_t *stream = new hipStream_t[num];

    // Иниацилизация потоков по одному потоку на каждое устройство
    if (!stream_init(stream))
        return 1;

    const int arraySize = 2;// 1024 * 1024 * 512;   
    const int sizePerSt = arraySize / num;

    /*if (!askAboutMemory(arraySize))
        return 2;*/

    float *a = new float[arraySize];
    float** dev_a;
    if (arraySize % num == 0)
    {
        // Предпологаем что количество эл-тов кратно количеству гпу
        // Проверить что сработает создавать массив из массивов для разных ГПУ
        dev_a = new float*[num];
        for (int j = 0; j < num; j++)
            dev_a[j] = new float[arraySize];
    }
    /*else
    {
        dev_a = new float* [num];
        for (int j = 0; j < num; j++)
            dev_a[j] = new float[sizePerSt];
    }   */

    if (!generateRandArray(a, arraySize))
        return 1;

    lazzzyArrayPrint(a, arraySize);

    //Подготовка к запуску ядра
    dim3 threads = dim3(2);
    dim3 blocks = dim3(1);

    // Подготовка и передач данных на карты
    try {
        for (int i = 0; i < num; i++)
        {
            hipSetDevice(i);
            hipMalloc((void**)&dev_a[i], arraySize * sizeof(float));
            hipMemcpyAsync(dev_a[i], a, arraySize * sizeof(float), hipMemcpyHostToDevice);
        }
    }
    catch (...) {
        return 3;
    }

    // Запуск ядра
    try {
        for (int i = 0; i < num; i++)
        {
            hipSetDevice(i);
            hipDeviceSynchronize();
            int d;
            hipGetDevice(&d);
            hipDeviceGetAttribute(&d, hipDeviceAttributePciDeviceId, d);

            printf("Kernel %d started\n", d);
            _kernel <<<blocks, threads, 0, stream[i]>>> (dev_a[i]);
            //printf("Kernel stoped\n");
        }
    }
    catch (...) {
        return 3;
    }

    // получение данных обратно
    try {
        for (int i = 0; i < num; i++)
        {
            delete a;
            hipSetDevice(i);            
            hipMemcpy(a, dev_a[i], arraySize * sizeof(float), hipMemcpyDeviceToHost);
            std::cout << "Theoretecly data from " << i << " device." << std::endl;
            lazzzyArrayPrint(a, arraySize);
        }
    }
    catch (...) {
        return 3;
    }

    hipFree(dev_a);

    //lazzzyArrayPrint(a, arraySize);

    // Убийство всех потоков
    if (!stream_dispose(stream))
        return 1;
    
    return 0;
}